#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
__global__ void add(int* a){ // Addition Kernel
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	a[tid]+=tid;
}
int main(int argc, char* argv[]){
	if(argc!=3){
		std::cout<<"Usage: "<<argv[0]<<" Numblocks BlockDim\n";
		return 0;
	}
	int nBlocks= atoi(argv[1]);
	int bDim = atoi(argv[2]);
	if(bDim>1024){
	std::cout<<"BlockDim should be less than or equal to 1024\n";
	return 0;
	}
	int* a = (int*) malloc (nBlocks*bDim*sizeof(int));// Allocate CPU Memory 
	for(int i=0;i<nBlocks*bDim;i++) // Initalize CPU array
		a[i]=i;
	int* da;
	hipMalloc(&da,nBlocks*bDim*sizeof(int)); // Initalize GPU array
	hipMemcpy(da,a,nBlocks*bDim*sizeof(int),hipMemcpyHostToDevice); // CPU->GPU

	add<<<nBlocks,bDim>>>(da); // Call addition kernel
	hipMemcpy(a,da,nBlocks*bDim*sizeof(int),hipMemcpyDeviceToHost); // GPU->CPU
	for(int i=0;i<nBlocks*bDim;i++) // Print final results
		std::cout<<a[i]<<"\n";
	free(a);
	hipFree(da);
	a=NULL;
	da=NULL;
	
}
