#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
__global__ void add(int* a){ // Addition Kernel
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	a[tid]+=tid;
}
int main(int argc, char* argv[]){
	if(argc!=3){
		std::cout<<"Usage: "<<argv[0]<<" Numblocks BlockDim\n";
		return 0;
	}
	int nBlocks= atoi(argv[1]);
	int bDim = atoi(argv[2]);
	if(bDim>1024){
	std::cout<<"BlockDim should be less than or equal to 1024\n";
	return 0;
	}
	int* da;
 	hipMallocManaged (&da, nBlocks*bDim*sizeof(int));// Allocate CPU/GPU Memory 
	for(int i=0;i<nBlocks*bDim;i++) // Initalize CPU array
		da[i]=i;
	add<<<nBlocks,bDim>>>(da); // Call addition kernel
	hipDeviceSynchronize();
	for(int i=0;i<nBlocks*bDim;i++) // Print final results
		std::cout<<da[i]<<"\n";
	hipFree(da);
	da=NULL;
	
}
