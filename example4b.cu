#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include"example.h"
__global__ void add(exampleStruct* example){// Addition Kernel
	exampleStruct my_example = example[blockIdx.x];
	my_example.index[threadIdx.x]+= my_example.base*blockDim.x+1000;
}
int main(int argc, char* argv[]){
	if(argc!=3){
		std::cout<<"Usage: "<<argv[0]<<" Numblocks BlockDim\n";
		return 0;
	}
	int nBlocks= atoi(argv[1]);
	int bDim = atoi(argv[2]);
	if(bDim>1024){
	std::cout<<"BlockDim should be less than or equal to 1024\n";
	return 0;
	}
	exampleStruct* example;
	hipMallocManaged(&example, nBlocks*sizeof(exampleStruct));// Allocate CPU Memory 
	for(int i=0;i<nBlocks;i++){// Initalize CPU array
		example[i].base=i;
		hipMallocManaged (&example[i].index,bDim*sizeof(int));
		for(int j=0;j<bDim;j++){
			example[i].index[j]=j;
		}
	}

	add<<<nBlocks,bDim>>>(example); // Call addition kernel
	// Error check
        hipError_t cudaerr1 = hipDeviceSynchronize();
                if (cudaerr1 != hipSuccess)
                        printf("kernel launch failed with error \"%s\".\n",
                                        hipGetErrorString(cudaerr1));
	
	for(int i=0;i<nBlocks;i++) // Print final results
		for(int j=0;j<bDim;j++)
			std::cout<<example[i].index[j]<<"\n";
	hipFree(example);
	example=NULL;
	
}
