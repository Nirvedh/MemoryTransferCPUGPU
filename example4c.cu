#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include"example.h"
__global__ void add(exampleStruct* example){// Addition Kernel
	exampleStruct my_example = example[blockIdx.x];
	my_example.index[threadIdx.x]+= my_example.base*blockDim.x+1000;
}
int main(int argc, char* argv[]){
	if(argc!=3){
		std::cout<<"Usage: "<<argv[0]<<" Numblocks BlockDim\n";
		return 0;
	}
	int nBlocks= atoi(argv[1]);
	int bDim = atoi(argv[2]);
	if(bDim>1024){
	std::cout<<"BlockDim should be less than or equal to 1024\n";
	return 0;
	}
	exampleStruct* example,*iexample,*dexample;
	example = (exampleStruct*) malloc (nBlocks*sizeof(exampleStruct));// Allocate CPU Memory 
	iexample = (exampleStruct*) malloc (nBlocks*sizeof(exampleStruct));// Allocate CPU Memory
	hipMalloc(&dexample,nBlocks*sizeof(exampleStruct));
	
	for(int i=0;i<nBlocks;i++){// Initalize CPU array
		example[i].base=i;
		example[i].index= (int*)malloc (bDim*sizeof(int));
		for(int j=0;j<bDim;j++){
			example[i].index[j]=j;
		}
	}
	// load into intermediate
	for(int i=0;i<nBlocks;i++){
		iexample[i].base=example[i].base;
		hipMalloc(&iexample[i].index,nBlocks*sizeof(exampleStruct));
		hipMemcpy(iexample[i].index,example[i].index,sizeof(exampleStruct),hipMemcpyHostToDevice);
	}
	
	hipMalloc(&dexample,nBlocks*sizeof(exampleStruct)); // Initalize GPU array
	hipMemcpy(dexample,iexample,nBlocks*sizeof(exampleStruct),hipMemcpyHostToDevice); // CPU->GPU

	add<<<nBlocks,bDim>>>(dexample); // Call addition kernel
	// Error check
        hipError_t cudaerr1 = hipDeviceSynchronize();
                if (cudaerr1 != hipSuccess)
                        printf("kernel launch failed with error \"%s\".\n",
                                        hipGetErrorString(cudaerr1));
	
	hipMemcpy(iexample,dexample,nBlocks*sizeof(exampleStruct),hipMemcpyDeviceToHost); // GPU->CPU
	// Copy back from intermediate
	for(int i=0;i<nBlocks;i++){
          	example[i].base=iexample[i].base;
                hipMemcpy(example[i].index,iexample[i].index,sizeof(exampleStruct),hipMemcpyDeviceToHost);
        }




	for(int i=0;i<nBlocks;i++) // Print final results
		for(int j=0;j<bDim;j++)
			std::cout<<example[i].index[j]<<"\n";
	free(example);
	hipFree(dexample);
	example=NULL;
	dexample=NULL;
	
}
